#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<string.h>

#define N 100
__global__ void comp(int *a,int *b,int *dot)
{
        int id=threadIdx.x;
        
        atomicAdd(out,(a[id]*b[id]));
}

int main()
{
        int a[N],b[N];
        int n,out;
        int *dev_a,*dev_b,*dev_c;

        hipMalloc((void **) &dev_a,N*sizeof(int));
        hipMalloc((void **) &dev_b,N*sizeof(int));
        hipMalloc((void **) &dev_c,sizeof(int));

        printf("\nEnter the number of elements:");
        scanf("%d",&n);
       printf("\nEnter array a\n")
       for(int i=0;i<n;i++)
       	scanf("%d",&a[i]);
      printf("\nEnter array b\n");
       for(int i=0;i<n;i++)
       	scanf("%d",&b[i]);
   

        hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(dev_b,b,N*sizeof(int),hipMemcpyHostToDevice);

        comp<<<1,n>>>(dev_a,dev_b,dev_c);

        hipMemcpy(&out,dev_c,sizeof(int),hipMemcpyDeviceToHost);

        printf("\n%d",out);
}
