#include "hip/hip_runtime.h"
# Cuda
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<string.h>

#define N 100
__global__ void comp(char *str1,char *str2,int len1,int len2,int *count)
{
        int id=threadIdx.x;
        int flag=0;
        for(int i=0;i<len2;i++,id++)
        {
                if(str1[id]!=str2[i])
                    flag=1;
        }
        if(flag==0)
               atomicAdd(count,1);

}

int main()
{
        char str1[N],str2[N];
        int len1,len2,count;
        char *dev_a,*dev_b;
        int *dev_c;

        hipMalloc((void **) &dev_a,N*sizeof(char));
        hipMalloc((void **) &dev_b,N*sizeof(char));
        hipMalloc((void **) &dev_c,sizeof(int));

        printf("\nEnter first string:");
        scanf("%s",str1);
        printf("\nEnter the substring:");
        scanf("%s",str2);
        len1=strlen(str1);
        len2=strlen(str2);

        hipMemcpy(dev_a,str1,N*sizeof(char),hipMemcpyHostToDevice);
        hipMemcpy(dev_b,str2,N*sizeof(char),hipMemcpyHostToDevice);

        comp<<<1,len1>>>(dev_a,dev_b,len1,len2,dev_c);

        hipMemcpy(&count,dev_c,sizeof(int),hipMemcpyDeviceToHost);

        printf("\n%d",count);
}
