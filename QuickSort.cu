#include "hip/hip_runtime.h"
#define N 5

using namespace std;

__global__ void quickSort(int *x, int *dfirst, int *dlast, int *list)
{
	int idx = threadIdx.x;
	int first = dfirst[idx];
	int last = dlast[idx];
	list[idx] = 0;

	if(first<last)
	{
		int pivot, j, temp, i;

		pivot = first;
		i = first;
		j = last;

		while(i<j)
		{
			while(x[i]<=x[pivot] && i<last)
				i++;
			while(x[j] > x[pivot])
				j--;
			if(i<j)
			{
				temp = x[i];
				x[i] = x[j];
				x[j] = temp;
			}
		}

		temp = x[pivot];
		x[pivot] = x[j];
		x[j] = temp;

		for(i=first; i<=last; i++)
			if(x[i] > x[i+1])
			{
				list[idx] = j+1;
				break;
			}
	}
}

int main()
{
	int a[N] = {1, 5, 9, 3, 6}, *da, i, size = N*sizeof(int), len = 0;
	int *list, *dlist, *dfirst, *dlast;

	hipMalloc(&da, size);
	hipMemcpy(da, a, size, hipMemcpyHostToDevice);

	vector<int> v;

	while(true)
	{
		size = (++len)*sizeof(int);

		int *first = (int *)malloc(size);
		int *last = (int *)malloc(size);

		first[0] = 0;
		last[len-1] = N-1;

		for(i=0; i<v.size(); i++)
		{
			first[i+1] = v[i]+1;
			last[i] = v[i]-1;
		}

		hipMalloc(&dfirst, size);
		hipMemcpy(dfirst, first, size, hipMemcpyHostToDevice);
		hipMalloc(&dlast, size);
		hipMemcpy(dlast, last, size, hipMemcpyHostToDevice);

		hipMalloc(&dlist, size);

		quickSort<<<1,len>>>(da, dfirst, dlast, dlist);

		list = (int *)malloc(size);
		hipMemcpy(list, dlist, size, hipMemcpyDeviceToHost);

		v.clear();
		for(i=0; i<len; i++)
			if(list[i] != 0)
				v.push_back(list[i]-1);
		len = v.size();

		if(len == 0)
			break;
	}

	hipMemcpy(a, da, N*sizeof(int), hipMemcpyDeviceToHost);
	for(i=0; i<N; i++)
		printf("%d\t", a[i]);
}
