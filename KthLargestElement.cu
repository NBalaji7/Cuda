#include "hip/hip_runtime.h"
__global__ void enumsort(int *deva, int *devn)
{
	int tid = threadIdx.x;
	int i, count=0;

	for(i=0;i<N;i++)
		if((deva[i]<=deva[tid])&&(i!=tid))
			count++;

	devn[count]=deva[tid];
}

int main(void)
{
	int a[] = {1, 5, 9, 3, 6};
	int *deva, *n, *devn;
	int i, k = 3;

	n = (int*)malloc(N*sizeof(int));

	hipMalloc((void**)&deva, N*sizeof(int));
	hipMalloc((void**)&devn, N*sizeof(int));
	hipMemcpy(deva, a, N*sizeof(int), hipMemcpyHostToDevice);

	enumsort<<<1,N>>>(deva,devn);

	hipMemcpy(n, devn, N*sizeof(int), hipMemcpyDeviceToHost);

	printf("\nSorted Array: \n");
	for(i=0;i<N;i++)
		printf("%d\t",n[i]);

	printf("\nThe kth largest element is : %d\n",n[N-k]);

	return 0;
}
