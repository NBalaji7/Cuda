#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define N 3
#define BLOCK_DIM 3

__global__ void matrixAdd(int *a,int *b,int *c)
{
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	int row=blockIdx.y*blockDim.y+threadIdx.y;
	int index=col+row*N;
	printf("\n%d\t%d",threadIdx.x,threadIdx.y);
	printf("\nIndex val:%d\n",index);
	if(col<N && row<N)
	{
		c[index]=a[index]+b[index];
	}
}

int main()
{
	int x[3][3]={1,2,3,4,5,6,7,8,9};
	int y[3][3]={0,1,2,3,4,5,6,7,8};
	int z[3][3];
	int i=0,j=0;
	int *dev_a,*dev_b,*dev_c;
	hipMalloc((void**)&dev_a,sizeof(x));
	hipMalloc((void**)&dev_b,sizeof(y));
	hipMalloc((void**)&dev_c,sizeof(z));
	hipMemcpy(dev_a,x,sizeof(x),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,y,sizeof(y),hipMemcpyHostToDevice);
	hipMemcpy(dev_c,z,sizeof(z),hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_DIM,BLOCK_DIM	);
	dim3 dimGrid((int)ceil(N/dimBlock.x),(int)ceil(N/dimBlock.y));
	matrixAdd<<<dimGrid,dimBlock>>>(dev_a,dev_b,dev_c);
	hipMemcpy(z,dev_c,sizeof(z),hipMemcpyDeviceToHost);
	printf("\noutput\n");
	for(i=0;i<3;i++)
	{
		for(j=0;j<3;j++)
		{
			printf("\n%d",z[i][j]);	
		}
	}
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}
